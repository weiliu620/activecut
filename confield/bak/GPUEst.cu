#include "hip/hip_runtime.h"
#include "common.h"

__global__ void EstKernel(const float* C, 
			  const unsigned int* gpuFwdMap1,
			  const unsigned int* gpuFwdMap2,
			  const unsigned short* gpuBwdMap1,
			  const unsigned short* gpuBwdMap2,
			  paraType par,
			  unsigned int N1,
			  unsigned int N2,
			  dim3 size,
			  float* gpuSumMat);

void GPUEst(const float* C,
	    const unsigned int* fwdMap1,
	    const unsigned int* fwdMap2,
	    const unsigned short* bwdMap1,
	    const unsigned short* bwdMap2,
	    paraType* par,
	    unsigned int N1,
	    unsigned int N2,
	    const unsigned short* size)
{
     double Jacob = 0, Jacob_alpha = 0;
     double DELL = 0, DELL_alpha = 0;
     double step = 1e6;
     double step_alpha = 1e6;
     unsigned int n = 0, n1 = 0, n2 = 0;
     double nsum = 0;
     unsigned int mapSize = size[0]*size[1]*size[2];
     // define simSize as argument of kernel fun.
     dim3 dimSize(size[0], size[1], size[2]);
     // create nsum on host. to save results from gpu.
     float* sumMat = (float*) malloc(N1*N2*sizeof(float));

     // pointer to device memory.
     float* gpuC;
     unsigned int* gpuFwdMap1;
     unsigned int* gpuFwdMap2;
     unsigned short* gpuBwdMap1;
     unsigned short* gpuBwdMap2;
     float* gpuSumMat;

     /* create input and output array on GPU. */
     hipMalloc((void**) &gpuC, sizeof(float)*N1*N2);
     checkCUDAError("GPUEst, allocate gpuC.");

     hipMalloc((void**) &gpuFwdMap1, sizeof(unsigned int)*mapSize);
     checkCUDAError("GPUEst, allocate fwdMap1");     
     hipMalloc((void**) &gpuFwdMap2, sizeof(unsigned int)*mapSize);
     checkCUDAError("GPUEst, allocate fwdMap2");     
     hipMalloc((void**) &gpuBwdMap1, sizeof(unsigned short)*N1*3);
     checkCUDAError("GPUEst, allocate bwdMap1");     
     hipMalloc((void**) &gpuBwdMap2, sizeof(unsigned short)*N2*3);
     checkCUDAError("GPUEst, allocate bwdMap2");     
     hipMalloc((void**) &gpuSumMat, sizeof(float)*N1*N2);
     checkCUDAError("GPUEst, allocate gpuSumMat.");

     /* host to device memory. */
     hipMemcpy(gpuC, C, sizeof(float)*N1*N2, hipMemcpyHostToDevice);
     checkCUDAError("GPUEst, memcpy gpuC");

     hipMemcpy(gpuFwdMap1, fwdMap1, sizeof(unsigned int)*mapSize, 
		hipMemcpyHostToDevice);
     checkCUDAError("GPUEst, memcpy gpuFwdMap1");
     hipMemcpy(gpuFwdMap2, fwdMap2, sizeof(unsigned int)*mapSize, 
		hipMemcpyHostToDevice);
     checkCUDAError("GPUEst, memcpy gpuFwdMap2");

     hipMemcpy(gpuBwdMap1, bwdMap1, sizeof(unsigned short)*N1*3, 
		hipMemcpyHostToDevice);
     checkCUDAError("GPUEst, memcpy gpuBwdMap1");
     hipMemcpy(gpuBwdMap2, bwdMap2, sizeof(unsigned short)*N2*3, 
		hipMemcpyHostToDevice);
     checkCUDAError("GPUEst, memcpy gpuBwdMap2");
     hipMemcpy(gpuSumMat, sumMat,  sizeof(float)*N1*N2, 
		hipMemcpyHostToDevice);
     checkCUDAError("GPUEst, memcpy gpuSumMat");

     /* run the kernel function. */
     int gridDimx = N1/BLOCK_SIZE_X + (N1%BLOCK_SIZE_X == 0?0:1);
     int gridDimy = N2/BLOCK_SIZE_Y + (N2%BLOCK_SIZE_Y == 0?0:1);

     dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
     dim3 dimGrid(gridDimx, gridDimy);
     if (_DBG >= 1){
	  printf("GPUEst, block size: %dx%d. gridsize: %dx%d\n", BLOCK_SIZE_X, BLOCK_SIZE_Y, gridDimx, gridDimy);
     }
     // call kernel, only to compute sum x_j for each x_i.
     EstKernel<<<dimGrid, dimBlock>>>(gpuC, gpuFwdMap1, gpuFwdMap2,
				      gpuBwdMap1, gpuBwdMap2, *par, N1, N2,
				      dimSize, gpuSumMat);
     hipMemcpy(sumMat, gpuSumMat, sizeof(float)*N1*N2, hipMemcpyDeviceToHost);

     // estimate alpha and beta by Newton's method.
     // estimate alpha
     float alpha_temp = par->alpha;
     while((abs(step_alpha) > CUEPS) && (abs(alpha_temp * 2) < MAXEXP)){
	  Jacob_alpha = 0;
	  DELL_alpha = 0;
	  for (n1 = 0; n1 < N1; n1++){
	       for (n2 = 0; n2 < N2; n2++) {
		    if (n1 != n2) {
			 nsum = sumMat[n1*N2+n2];
			 // 2nd derivative.
			 Jacob_alpha = Jacob_alpha + 
			      tanh(alpha_temp + par->beta*nsum)*tanh(alpha_temp + par->beta*nsum) - 1;
			 DELL_alpha = DELL_alpha + C[n1*N2+n2] - tanh(alpha_temp + par->beta*nsum);
		    }
	       }
	  }
	  if (Jacob_alpha != 0) {
	       step_alpha = -DELL_alpha/Jacob_alpha;
	       alpha_temp = alpha_temp + step_alpha;
	       printf("GPUEst.cu: alpha = %f\n", par->alpha);
	  }
	  else {
	       fprintf(stderr, "GPUEst.cu: DELL_alpha zero. alpha does not change.");
	  }
     }
     par->alpha = alpha_temp;

     // estimate beta.
     while((abs(step) > CUEPS) && (par->beta < CUMAXBETA)){
	  Jacob = 0;
	  DELL = 0;
	  for (n1 = 0; n1 < N1; n1++){
	       for (n2 = 0; n2 < N2; n2++) {
		    if (n1 != n2) {
			 nsum = sumMat[n1*N2+n2];
			 // 2nd derivative.
			 Jacob = Jacob + 
			      nsum * nsum * (tanh(par->beta*nsum)*tanh(par->alpha + par->beta*nsum) - 1);
			 DELL = DELL + nsum * (C[n1*N2+n2] - tanh(par->alpha + par->beta*nsum));
		    }
	       }
	  }
	  if (Jacob != 0){
	       step = -DELL/Jacob;
	       par->beta = par->beta + step;
	       if (par->beta >= CUMAXBETA){
		    printf("GPUEst.cu: beta too large. Set Beta to largest value.\n");
		    par->beta = CUMAXBETA;
	       }
	  }
	  else{
	       fprintf(stderr, "GPUEst.cu: DELL zero. Netwon's method deviced by zero.\n");
	       printf("GPUEst.cu: jacobian is zero. Probably newBeta is too large, \n so we just set beta to largest value.\n");
	       par->beta = CUMAXBETA;
	  }
	  if (_DBG >= 1){
	       printf("GPUEst.cu: beta = %f\n", par->beta);
	  }

     }
     if (_DBG >= 1){
	  printf("GPUEst.cu: beta = %f\n", par->beta);
     }



     /* clean up. */
     hipFree(gpuC);
     hipFree(gpuFwdMap1);
     hipFree(gpuFwdMap2);
     hipFree(gpuBwdMap1);
     hipFree(gpuBwdMap2);
     hipFree(gpuSumMat);
     free(sumMat);

}

/* Kernel function  */
__global__ void EstKernel(const float* C, 
			  const unsigned int* gpuFwdMap1,
			  const unsigned int* gpuFwdMap2,
			  const unsigned short* gpuBwdMap1,
			  const unsigned short* gpuBwdMap2,
			  paraType par,
			  unsigned int N1,
			  unsigned int N2,
			  dim3 size,
			  float* gpuSumMat)

{     
     unsigned short i;
     unsigned short j;
     unsigned short k;
     int di, dj, dk;
     int ti, tj, tk;
     uint n;
     uint n1 = blockIdx.x*blockDim.x + threadIdx.x;
     uint n2 = blockIdx.y*blockDim.y + threadIdx.y;

     float gc1 = 1/(sqrt(2*PI*par.sigma21));
     float gc2 = 1/(sqrt(2*PI*par.sigma22));
     float sum = 0;

     // thread fall outside of matrix C, or mask is zero.
     if (n1 >= N1 | n2 >= N2) {return;}

     // image one's neighbors.
     i = gpuBwdMap1[n1*3 + 0];
     j = gpuBwdMap1[n1*3 + 1];
     k = gpuBwdMap1[n1*3 + 2];

     for (di = -1; di <= 1; di ++){
	  for (dj = -1; dj <= 1; dj ++){
	       for (dk = -1; dk <= 1; dk ++){
		    ti = i + di;
		    tj = j + dj;
		    tk = k + dk;
		    if ((ti >= 0 && ti < size.x
			 && tj >= 0 && tj < size.y
			 && tk >= 0 && tk < size.z)
			 && (gpuFwdMap1[ti * size.y*size.z +  tj * size.z + tk] > 0)){
		    n = gpuFwdMap1[ti * size.y * size.z +  tj * size.z + tk];
		    sum = sum + C[n*N2 + n2];
		    }
	       }
	  }
     }
     sum = sum - C[n1*N2 + n2];
     // image 2's neighbors.
     i = gpuBwdMap2[n2*3 + 0];
     j = gpuBwdMap2[n2*3 + 1];
     k = gpuBwdMap2[n2*3 + 2];
     for (di = -1; di <= 1; di ++){
	  for (dj = -1; dj <= 1; dj ++){
	       for (dk = -1; dk <= 1; dk ++){
		    ti = i + di;
		    tj = j + dj;
		    tk = k + dk;
		    if ((ti >= 0 && ti < size.x
			 && tj >= 0 && tj < size.y
			 && tk >= 0 && tk < size.z)
			 && (gpuFwdMap2[ti * size.y*size.z +  tj * size.z + tk] > 0)){
		    n = gpuFwdMap2[ti * size.y * size.z +  tj * size.z + tk];
		    sum = sum + C[n1*N2 + n];
		    }
	       }
	  }
     }
     sum = sum - C[n1*N2 + n2];
     
     gpuSumMat[n1*N2+n2] = sum;
#if __DEVICE_EMULATION__
     if (n1 == 0 && n2 == 0){

     }
#endif


}
