#include "hip/hip_runtime.h"
#include "common.h"

__global__ void EstKernel(const float* C, 
			  const unsigned int* gpuFwdMap1,
			  const unsigned int* gpuFwdMap2,
			  const unsigned short* gpuBwdMap1,
			  const unsigned short* gpuBwdMap2,
			  paraType par,
			  unsigned int N1,
			  unsigned int N2,
			  dim3 size,
			  float* gpuSumMat);

void GPUEst(const float* C,
	    const unsigned int* fwdMap1,
	    const unsigned int* fwdMap2,
	    const unsigned short* bwdMap1,
	    const unsigned short* bwdMap2,
	    paraType* par,
	    unsigned int N1,
	    unsigned int N2,
	    const unsigned short* size)
{
     double Jacob = 0;
     double DELL = 0;
     double step = 1e6;
     unsigned int n = 0;
     double nsum = 0;
     unsigned int mapSize = size[0]*size[1]*size[2];
     // define simSize as argument of kernel fun.
     dim3 dimSize(size[0], size[1], size[2]);
     // create nsum on host. to save results from gpu.
     float* sumMat = (float*) malloc(N1*N2*sizeof(float));

     // pointer to device memory.
     float* gpuC;
     unsigned int* gpuFwdMap1;
     unsigned int* gpuFwdMap2;
     unsigned short* gpuBwdMap1;
     unsigned short* gpuBwdMap2;
     float* gpuSumMat;

     /* create input and output array on GPU. */
     hipMalloc((void**) &gpuC, sizeof(float)*N1*N2);
     checkCUDAError("GPUEst, allocate gpuC.");

     hipMalloc((void**) &gpuFwdMap1, sizeof(unsigned int)*mapSize);
     checkCUDAError("GPUEst, allocate fwdMap1");     
     hipMalloc((void**) &gpuFwdMap2, sizeof(unsigned int)*mapSize);
     checkCUDAError("GPUEst, allocate fwdMap2");     
     hipMalloc((void**) &gpuBwdMap1, sizeof(unsigned short)*N1*3);
     checkCUDAError("GPUEst, allocate bwdMap1");     
     hipMalloc((void**) &gpuBwdMap2, sizeof(unsigned short)*N2*3);
     checkCUDAError("GPUEst, allocate bwdMap2");     
     hipMalloc((void**) &gpuSumMat, sizeof(float)*N1*N2);
     checkCUDAError("GPUEst, allocate gpuSumMat.");

     /* host to device memory. */
     hipMemcpy(gpuC, C, sizeof(float)*N1*N2, hipMemcpyHostToDevice);
     checkCUDAError("GPUEst, memcpy gpuC");

     hipMemcpy(gpuFwdMap1, fwdMap1, sizeof(unsigned int)*mapSize, 
		hipMemcpyHostToDevice);
     checkCUDAError("GPUEst, memcpy gpuFwdMap1");
     hipMemcpy(gpuFwdMap2, fwdMap2, sizeof(unsigned int)*mapSize, 
		hipMemcpyHostToDevice);
     checkCUDAError("GPUEst, memcpy gpuFwdMap2");

     hipMemcpy(gpuBwdMap1, bwdMap1, sizeof(unsigned short)*N1*3, 
		hipMemcpyHostToDevice);
     checkCUDAError("GPUEst, memcpy gpuBwdMap1");
     hipMemcpy(gpuBwdMap2, bwdMap2, sizeof(unsigned short)*N2*3, 
		hipMemcpyHostToDevice);
     checkCUDAError("GPUEst, memcpy gpuBwdMap2");
     hipMemcpy(gpuSumMat, sumMat,  sizeof(float)*N1*N2, 
		hipMemcpyHostToDevice);
     checkCUDAError("GPUEst, memcpy gpuSumMat");

     /* run the kernel function. */
     int gridDimx = N1/BLOCK_SIZE_X + (N1%BLOCK_SIZE_X == 0?0:1);
     int gridDimy = N2/BLOCK_SIZE_Y + (N2%BLOCK_SIZE_Y == 0?0:1);

     dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
     dim3 dimGrid(gridDimx, gridDimy);
     if (_DBG >= 1){
	  printf("GPUEst, block size: %dx%d\n", BLOCK_SIZE_X, BLOCK_SIZE_Y);
	  printf("GPUEst, gridsize: %dx%d\n", gridDimx, gridDimy);
     }

     // estimate beta by Newton's method.
     while((step > CUEPS) && (par->beta < CUMAXBETA)){
	  // call kernel, only to compute sum x_j for each x_i.
	  EstKernel<<<dimGrid, dimBlock>>>(gpuC, gpuFwdMap1, gpuFwdMap2,
					   gpuBwdMap1, gpuBwdMap2, *par, N1, N2,
					   dimSize, gpuSumMat);
	  hipMemcpy(sumMat, gpuSumMat, sizeof(float)*N1*N2, hipMemcpyDeviceToHost);

	  Jacob = 0;
	  DELL = 0;
	  for (n = 0; n < N1*N2; n++){
	       nsum = sumMat[n];
	       // 2nd derivative.
	       Jacob = Jacob + 
		    nsum * nsum * (tanh(par->beta*nsum)*tanh(par->beta*nsum) - 1);
	       DELL = DELL + nsum * (C[n] - tanh(par->beta*nsum));
	  }
	  if (Jacob != 0){
	       step = -DELL/Jacob;
	       par->beta = par->beta + step;
	       if (par->beta >= CUMAXBETA){
		    printf("GPUEst.cu: beta too large. Set Beta to largest value.\n");
		    par->beta = CUMAXBETA;
	       }
	  }
	  else{
	       fprintf(stderr, "GPUEst.cu: DELL zero. Netwon's method deviced by zero.\n");
	       printf("GPUEst.cu: jacobian is zero. Probably newBeta is too large, \n so we just set beta to largest value.\n");
	       par->beta = CUMAXBETA;
	  }
	  if (_DBG >= 1){
	       printf("GPUEst.cu: beta = %f\n", par->beta);
	  }

     }
     if (_DBG >= 1){
	  printf("GPUEst.cu: beta = %f\n", par->beta);
     }

     /* clean up. */
     hipFree(gpuC);
     hipFree(gpuFwdMap1);
     hipFree(gpuFwdMap2);
     hipFree(gpuBwdMap1);
     hipFree(gpuBwdMap2);
     hipFree(gpuSumMat);
     free(sumMat);

}

/* Kernel function  */
__global__ void EstKernel(const float* C, 
			  const unsigned int* gpuFwdMap1,
			  const unsigned int* gpuFwdMap2,
			  const unsigned short* gpuBwdMap1,
			  const unsigned short* gpuBwdMap2,
			  paraType par,
			  unsigned int N1,
			  unsigned int N2,
			  dim3 size,
			  float* gpuSumMat)

{     
     unsigned short i;
     unsigned short j;
     unsigned short k;
     int di, dj, dk;
     int ti, tj, tk;
     uint n;
     uint n1 = blockIdx.x*blockDim.x + threadIdx.x;
     uint n2 = blockIdx.y*blockDim.y + threadIdx.y;

     float gc1 = 1/(sqrt(2*PI*par.sigma21));
     float gc2 = 1/(sqrt(2*PI*par.sigma22));
     float sum = 0;

     // thread fall outside of matrix C, or mask is zero.
     if (n1 >= N1 | n2 >= N2) {return;}

     // image one's neighbors.
     i = gpuBwdMap1[n1*3 + 0];
     j = gpuBwdMap1[n1*3 + 1];
     k = gpuBwdMap1[n1*3 + 2];

     for (di = -1; di <= 1; di ++){
	  for (dj = -1; dj <= 1; dj ++){
	       for (dk = -1; dk <= 1; dk ++){
		    ti = i + di;
		    tj = j + dj;
		    tk = k + dk;
		    if ((ti >= 0 && ti < size.x
			 && tj >= 0 && tj < size.y
			 && tk >= 0 && tk < size.z)
			 && (gpuFwdMap1[ti * size.y*size.z +  tj * size.z + tk] > 0)){
		    n = gpuFwdMap1[ti * size.y * size.z +  tj * size.z + tk];
		    sum = sum + C[n*N2 + n2];
		    }
	       }
	  }
     }
     sum = sum - C[n1*N2 + n2];
     // image 2's neighbors.
     i = gpuBwdMap2[n2*3 + 0];
     j = gpuBwdMap2[n2*3 + 1];
     k = gpuBwdMap2[n2*3 + 2];
     for (di = -1; di <= 1; di ++){
	  for (dj = -1; dj <= 1; dj ++){
	       for (dk = -1; dk <= 1; dk ++){
		    ti = i + di;
		    tj = j + dj;
		    tk = k + dk;
		    if ((ti >= 0 && ti < size.x
			 && tj >= 0 && tj < size.y
			 && tk >= 0 && tk < size.z)
			 && (gpuFwdMap2[ti * size.y*size.z +  tj * size.z + tk] > 0)){
		    n = gpuFwdMap2[ti * size.y * size.z +  tj * size.z + tk];
		    sum = sum + C[n1*N2 + n];
		    }
	       }
	  }
     }
     sum = sum - C[n1*N2 + n2];
     
     gpuSumMat[n1*N2+n2] = sum;
#if __DEVICE_EMULATION__
     if (n1 == 0 && n2 == 0){

     }
#endif


}
